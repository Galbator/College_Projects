#include <hip/hip_runtime.h>
#include <cstdio>

int main(int argc, char *argv[]) {
    int driver_version = 0;
    int runtime_version = 0;

    hipError_t err1 = hipDriverGetVersion(&driver_version);
    hipError_t err2 = hipRuntimeGetVersion(&runtime_version);
    if (err1 != hipSuccess) {
        printf("err1 %s \n", hipGetErrorString(err1));
    }
    if (err2 != hipSuccess) {
        printf("err2 %s \n", hipGetErrorString(err2));
    }

    printf("Driver version: %d, Runtime version: %d\n",
            driver_version, runtime_version);
    
    printf("Driver API version: %d\n", CUDA_VERSION);
}

