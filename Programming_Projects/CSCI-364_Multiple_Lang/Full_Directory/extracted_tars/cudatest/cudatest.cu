#include <cstdio>

void printProperties(int);

int main(int argc, char *argv[]) {
    int nDevices = 0;
    hipError_t err = hipGetDeviceCount(&nDevices);

    if (err != hipSuccess) {
        printf("%s\n", hipGetErrorString(err));
    } else {
        printf("Number of devices: %d\n", nDevices);
    }

    for (int i = 0; i < nDevices; i++) {
        printProperties(i);
    }
}

void printProperties(int i) {
    struct hipDeviceProp_t cdp;
    hipGetDeviceProperties(&cdp, i);

    printf("\nCUDA Device Properties - Device: %d\n", i);
    printf("----------------------------------\n");
    printf("  Device name: %s\n", cdp.name);
    printf("  Compute capability: %d.%d\n", cdp.major, cdp.minor);
	printf("  Total global memory: %lu bytes\n", cdp.totalGlobalMem);
    printf("  Total constant memory: %lu bytes\n", cdp.totalConstMem);
	printf("  Max grid size, dim(0): %d\n", cdp.maxGridSize[0]);
	printf("  Max grid size, dim(1): %d\n", cdp.maxGridSize[1]);
	printf("  Max grid size, dim(2): %d\n", cdp.maxGridSize[2]);
	printf("  Max threads per block: %d\n", cdp.maxThreadsPerBlock);
	printf("  Max block size, dim(0): %d\n", cdp.maxThreadsDim[0]);
	printf("  Max block size, dim(1): %d\n", cdp.maxThreadsDim[1]);
	printf("  Max block size, dim(2): %d\n", cdp.maxThreadsDim[2]);
	printf("  Shared memory per block: %lu bytes\n", 
            cdp.sharedMemPerBlock);
	printf("  Registers per block: %d\n", cdp.regsPerBlock);
	printf("  Clock frequency: %d khz\n", cdp.clockRate);
	printf("  Asynchronous engines: %d\n", cdp.asyncEngineCount);
	printf("  Multiprocessors on device: %d\n", 
            cdp.multiProcessorCount);
    //printf("  CUDA Cores/MP: %3d\n", _ConvertSMVer2Cores(cdp.major, cdp.minor));
    printf("  Concurrent kernels: %s\n", cdp.concurrentKernels ? "yes" : "no");
    printf("  Compute mode: %d\n", cdp.computeMode);
    printf("and still more properties...\n");
}
