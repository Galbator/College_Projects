// Mason Motschke - CSCI 364


#include <hip/hip_runtime.h>
#include <algorithm>    // min, max
#include <fstream>
#include <iostream>
#include <cstdio>
#include <cstdlib>      // exit
#include <string>

void usage(char *name) {
    printf("Usage: %s <bw output file> <rgb input file> <num rows> <num cols>\n", name);
}

void load_rgbfile(std::string filename, int *rgb_data, int height, int width) {
    std::ifstream ifs(filename, std::ifstream::in);
    int red, green, blue;
    int rgb_index;

    for (int row = 0; row < height; row++) {
        for (int col = 0; col < width; col++) {
            // read the channel values
            ifs >> red >> green >> blue;

            // store in array
            rgb_index = 3 * row * width + col;
            rgb_data[rgb_index] = red;
            rgb_data[rgb_index + 1] = green;
            rgb_data[rgb_index + 2] = blue;
        }
    }

    ifs.close();
}

void convert_avg(int *in, int *out, int height, int width) {
    int rgb_index;
    int red, green, blue;
    int grey_index;
    int grey;

    for (int row = 0; row < height; row++) {
        for (int col = 0; col < width; col++) {
            rgb_index = 3 * row * width + col;

            red = in[rgb_index];
            green = in[rgb_index + 1];
            blue = in[rgb_index + 2];

            grey = (red + green + blue) / 3;
            grey_index = row * width + col;
            out[grey_index] = grey;
        }
    }
}

__global__ void device_av(int n, int *rgb, int *bw, int width){
  //printf("Hello\n");
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n){
    for (int col = 0; col < width; col++) {
      int rgb_index = 3 * i * width + col;

      int red = rgb[rgb_index];
      int green = rgb[rgb_index + 1];
      int blue = rgb[rgb_index + 2];

      int grey = (red + green + blue) / 3;
      int grey_index = i * width + col;
      bw[grey_index] = grey;
    }    
  }
}

void write_bwfile(std::string filename, int *bw_data, int height, int width) {
    std::ofstream ofs(filename, std::ofstream::out);

    int grey_index;
    for (int row = 0; row < height; row++) {
        for (int col = 0; col < width; col++) {
            grey_index = row * width + col;
            ofs << bw_data[grey_index] << " ";
        }
        ofs << "\n";
    }
    ofs << "\n";

    ofs.close();
}

int main(int argc, char *argv[]) {
    using std::cout;
    using std::endl;

    // check command line args count
    if (argc != 5) {
        usage(argv[0]);
        exit(1);
    }

    // get values from the command line
    std::string outfile = argv[1];
    std::string infile = argv[2];
    const int HEIGHT = std::stoi(argv[3]);
    const int WIDTH = std::stoi(argv[4]);

    cout << "Output file: " << outfile << endl;
    cout << "Input file: " << infile << endl;
    cout << "HEIGHT: " << HEIGHT << endl;
    cout << "WIDTH: " << WIDTH << endl;

    // create data structures

    // - GPU
    int n = HEIGHT;

    int *rgb_data = new int[HEIGHT * WIDTH * 3];        // 3 the for red, green, and blue values
    int *bw_data = new int[HEIGHT * WIDTH];

    // load rgb data from file
    load_rgbfile(infile, rgb_data, HEIGHT, WIDTH);
    
    // - GPU
    int *drgb, *dbw;
    hipMalloc(&drgb, sizeof(int) * HEIGHT * WIDTH * 3);
    hipMalloc(&dbw, sizeof(int) * HEIGHT * WIDTH);

    hipMemcpy(drgb, rgb_data, sizeof(int) * HEIGHT * WIDTH * 3, hipMemcpyHostToDevice);

    // convert rgb to greyscale using the "average" method
    //convert_avg(rgb_data, bw_data, HEIGHT, WIDTH);

    device_av<<<32, 94>>>(n, drgb, dbw, WIDTH);
    hipDeviceSynchronize();

    // write greyscale data to file
    hipMemcpy(bw_data, dbw, sizeof(int) * HEIGHT * WIDTH, hipMemcpyDeviceToHost);
    write_bwfile(outfile, bw_data, HEIGHT, WIDTH);

    // free memory

    // - GPU
    hipFree(drgb);
    hipFree(dbw);

    delete []rgb_data;
    delete []bw_data;
}
