
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void parallel() {
  printf("This is running in parallel.\n");
}

int main() {
  dim3 dimGrid(2, 2);
  dim3 dimBlock(2, 2, 2);

  parallel<<<dimGrid, dimBlock>>>();
  hipDeviceSynchronize();
}

