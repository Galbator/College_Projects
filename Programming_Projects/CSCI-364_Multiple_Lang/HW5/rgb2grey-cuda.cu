
#include <hip/hip_runtime.h>
#include <algorithm>    // min, max
#include <fstream>
#include <iostream>
#include <cstdio>
#include <cstdlib>      // exit
#include <string>

void usage(char *name) {
    printf("Usage: %s <bw output file> <rgb input file> <num rows> <num cols>\n", name);
}

void load_rgbfile(std::string filename, int *rgb_data, int height, int width) {
    std::ifstream ifs(filename, std::ifstream::in);
    int red, green, blue;
    int rgb_index;

    for (int row = 0; row < height; row++) {
        for (int col = 0; col < width; col++) {
            // read the channel values
            ifs >> red >> green >> blue;

            // store in array
            rgb_index = 3 * row * width + col;
            rgb_data[rgb_index] = red;
            rgb_data[rgb_index + 1] = green;
            rgb_data[rgb_index + 2] = blue;
        }
    }

    ifs.close();
}

void convert_avg(int *in, int *out, int height, int width) {
    int rgb_index;
    int red, green, blue;
    int grey_index;
    int grey;

    for (int row = 0; row < height; row++) {
        for (int col = 0; col < width; col++) {
            rgb_index = 3 * row * width + col;

            red = in[rgb_index];
            green = in[rgb_index + 1];
            blue = in[rgb_index + 2];

            grey = (red + green + blue) / 3;
            grey_index = row * width + col;
            out[grey_index] = grey;
        }
    }
}

void write_bwfile(std::string filename, int *bw_data, int height, int width) {
    std::ofstream ofs(filename, std::ofstream::out);

    int grey_index;
    for (int row = 0; row < height; row++) {
        for (int col = 0; col < width; col++) {
            grey_index = row * width + col;
            ofs << bw_data[grey_index] << " ";
        }
        ofs << "\n";
    }
    ofs << "\n";

    ofs.close();
}

int main(int argc, char *argv[]) {
    using std::cout;
    using std::endl;

    // check command line args count
    if (argc != 5) {
        usage(argv[0]);
        exit(1);
    }

    // get values from the command line
    std::string outfile = argv[1];
    std::string infile = argv[2];
    const int HEIGHT = std::stoi(argv[3]);
    const int WIDTH = std::stoi(argv[4]);

    cout << "Output file: " << outfile << endl;
    cout << "Input file: " << infile << endl;
    cout << "HEIGHT: " << HEIGHT << endl;
    cout << "WIDTH: " << WIDTH << endl;

    // create data structures
    int *rgb_data = new int[HEIGHT * WIDTH * 3];        // 3 the for red, green, and blue values
    int *bw_data = new int[HEIGHT * WIDTH];

    // load rgb data from file
    load_rgbfile(infile, rgb_data, HEIGHT, WIDTH);

    // convert rgb to greyscale using the "average" method
    convert_avg(rgb_data, bw_data, HEIGHT, WIDTH);

    // write greyscale data to file
    write_bwfile(outfile, bw_data, HEIGHT, WIDTH);

    // free memory
    delete []rgb_data;
    delete []bw_data;
}
